
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <map>
#include <string>
#include <stdio.h>

using namespace std;
__global__
void divide(int N, int* m){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    for(int z = index; z < N; z += stride)
    {
	    m[z] = 0;
	    for(int id = 1; id < z; id++){
		if(z%id == 0){
		    m[z] +=id;
		}
	    }
    }
}

int main(int argc, char** argv){
    const int k = 1000000;
    int *m;
    hipMallocManaged(&m, k*sizeof(int));
    int blockSize=  256;
    int numBlocks = (k + blockSize - 1) / blockSize;
    divide<<< numBlocks, blockSize>>> (k, m);

    hipDeviceSynchronize();

    cout <<"\n";
    int res = 0;
    for(int i = 1; i < k; i++){
        int val = m[i];
        if(val < k && val != i && m[val] == i){
            res += i;
            res += val;
	    m[i]=0;
	    m[val]=0;
        }
    }
    cout << "res: "<< res << "\n";
    hipFree(m);
    return 0;
}
